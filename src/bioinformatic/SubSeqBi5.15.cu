#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h> 

/*

Versão do Daniel com memória compartilhada e com 2 gpus
Melhor versão

*/

#define N_COL 2048 
#define N_LIN 2048

#define BLOCK_SIZE_SOMA_PREF 2
#define N_THREAD_PER_BLOCK_SOMA_PREF 1024

#define BLOCK_SIZE_CALC 16 
#define N_THREAD_PER_BLOCK_CALC 1024

#define N_GPU 1

__global__ void somaPrex(int* matriz, int nColuna, int nLin){ 

   int idThread = blockIdx.x * blockDim.x + threadIdx.x;

   int qtdColSoma = (nLin / (blockDim.x * gridDim.x));

   int comecDeTrabThr = ((qtdColSoma * nColuna) * idThread); 

   int i,j;
   for(j=0; j<qtdColSoma;j++){
	   for(i=1; i<nColuna; i++){
		   matriz[(comecDeTrabThr + i) + (j*nColuna)] += matriz[(j*nColuna) + (comecDeTrabThr + i) - 1];
	   }
   }
}

__global__ void calc(int* matriz, int nCol, int nLin, int qtdDeParesGH,  int *Subseqs, int *auxMatriz, int GPU){

	long int idThread = (blockIdx.x * blockDim.x + threadIdx.x);
	int tidAux        = blockIdx.x * blockDim.x + threadIdx.x;

	__shared__ int vetComp[49152];

	int i,j,k,l,auxComp,comecoVetThreadComp;
	
	//      M         t_m       S      suf
	int ini_M, fim_M, t_M, ini_S, fim_S, suf;
	ini_M = fim_M = ini_S = fim_S = -1;
 
	comecoVetThreadComp = (threadIdx.x * 12);
		
	auxComp = t_M = suf = 0;	

	for(l=0; l< ((qtdDeParesGH/(blockDim.x * gridDim.x))+1); l++){
	
		 float delta = ((4*(nCol*nCol)) - (4*nCol) + 1) - (8 * (idThread+1));
		 float auxG = ((((2*nCol) - 3) - sqrt(delta))/2);

		 if(auxG - (int)auxG){
		    auxG += 1;
         }

		 long int g = auxG;

		 float auxH= (idThread+1) - (((((2*nCol)-1)-g) * g)/2)+g;
    	 long int h = auxH;

		 if(g>=0 && g<=nCol && h>=0 && h<=nCol && idThread < qtdDeParesGH){
			 if(g==0){
			   for(j=0; j < nLin; j+=12){
				   for(k=0;k<12;k++){
					   vetComp[(threadIdx.x * 12)+k] = matriz[(h + (nCol * (k+j)))+(nCol*GPU)]; 
					}

					for(i = (comecoVetThreadComp -1); i < (comecoVetThreadComp + 12)-1; i++){
						if(i == fim_M){
							fim_S++;
							suf += vetComp[i+1];

							if(suf < 0){
								suf = 0;
								fim_S = -1;
							}

							ini_S = fim_S == 0 ? 0 : ini_S; // Inicio S

							  if(vetComp[i+1] > 0){
								   fim_M++;
								   t_M += vetComp[i+1];
								   ini_M = fim_M == 0 ? 0 : ini_M; // Inicio M
							   }
						}
						else{
							 if(suf + vetComp[i+1] > t_M){
								 fim_S++;
								 if(ini_M == -1){
								  fim_S = ini_S = i +1;
								 }

								 suf += vetComp[i+1];
								 ini_M = ini_S;
								 fim_M = fim_S;
								 t_M = suf;
							 }
							 else{
								    if(suf + vetComp[i+1] > 0){
								        fim_S++;
								        if(suf == 0){
								            ini_S = fim_S = i+1;
								        }

								        suf += vetComp[i+1];
								    }
								    else{
								        ini_S = fim_S = i + 2;
								        suf = 0;
								    }
							 }
						}
					}
			   }

				if(t_M > auxComp){																									
		  		   Subseqs[tidAux] = t_M;
				   auxComp = t_M;
			    }

				idThread += (blockDim.x * gridDim.x);
			 }
			 else{
				 for(j=0; j < nLin; j+=12){
				   for(k=0;k<12;k++){
					   vetComp[(threadIdx.x * 12)+k]  = matriz[(h + (nLin * (k+j)))+(nCol*GPU)] - matriz[((g-1) + (nLin * (k+j)))+(nCol*GPU)];
					}

					for(i = (comecoVetThreadComp -1); i < (comecoVetThreadComp + 12)-1; i++){
						if(i == fim_M){
							fim_S++;
							suf += vetComp[i+1];

							if(suf < 0){
								suf = 0;
								fim_S = -1;
							}

							ini_S = fim_S == 0 ? 0 : ini_S; // Inicio S

							  if(vetComp[i+1] > 0){
								   fim_M++;
								   t_M += vetComp[i+1];
								   ini_M = fim_M == 0 ? 0 : ini_M; // Inicio M
							   }
						}
						else{
							 if(suf + vetComp[i+1] > t_M){
								 fim_S++;
								 if(ini_M == -1){
								  fim_S = ini_S = i +1;
								 }

								 suf += vetComp[i+1];
								 ini_M = ini_S;
								 fim_M = fim_S;
								 t_M = suf;
							 }
							 else{
								    if(suf + vetComp[i+1] > 0){
								        fim_S++;
								        if(suf == 0){
								            ini_S = fim_S = i+1;
								        }

								        suf += vetComp[i+1];
								    }
								    else{
								        ini_S = fim_S = i + 2;
								        suf = 0;
								    }
							 }
						}
					}
			     }

				  if(t_M > auxComp){																									
		  		     Subseqs[tidAux] = t_M;
				     auxComp = t_M;
			      }
				  
		 		  idThread += (blockDim.x * gridDim.x);	
			 }
		 }
	}
}

int main(){

   float elapsedTime;    // Tempo
   hipEvent_t start, stop; // Tempo

   int i; int qtdDeParesGH = (((N_COL/2)*((N_COL-1)/2)) / 2);
	
   printf("\n Pares G e H Calculados %d \n",qtdDeParesGH);

   //Alocando a matriz no host
   int *matriz_h = (int *)malloc(sizeof(int *) * (N_COL*N_LIN));
   int *subSeq_h = (int *)malloc(sizeof(int *) * ((BLOCK_SIZE_CALC*N_THREAD_PER_BLOCK_CALC)*N_GPU));
   int *matriz_d; int *subSeq_d; int *auxMatriz_d;

   //Preenchendo a matriz no host
   for(i=0; i<(N_COL*N_LIN); i++){
       matriz_h[i] = -1;
   }

   for(i=0; i<(BLOCK_SIZE_CALC*N_THREAD_PER_BLOCK_CALC); i++){
       subSeq_h[i] = 0;
   }

   matriz_h[228] = 580;   matriz_h[229] = 280;
		    

   for(i=0; i<N_GPU; i++){
		hipSetDevice(i);

   		//Reservando espaco na GPU
   		hipMalloc((void**)&matriz_d, (N_COL*N_LIN)  * sizeof(int)); 
   		hipMalloc((void**)&auxMatriz_d, ((BLOCK_SIZE_CALC*N_THREAD_PER_BLOCK_CALC) * N_LIN)  * sizeof(int)); 
   		hipMalloc((void**)&subSeq_d, (BLOCK_SIZE_CALC*N_THREAD_PER_BLOCK_CALC) * sizeof(int));

   		hipMemcpy(matriz_d, matriz_h, (N_COL*N_LIN) * sizeof(int), hipMemcpyHostToDevice);

	}

   hipEventCreate(&start); // Contagem do tempo
   hipEventCreate(&stop);
   hipEventRecord(start, 0);

   for(i=0; i<N_GPU; i++){
		hipSetDevice(i);
	    somaPrex<<<BLOCK_SIZE_SOMA_PREF, N_THREAD_PER_BLOCK_SOMA_PREF>>>(matriz_d,N_COL,N_LIN); 
   }

   hipDeviceSynchronize();

   for(i=0; i<N_GPU; i++){
      calc<<<BLOCK_SIZE_CALC, N_THREAD_PER_BLOCK_CALC>>>(matriz_d,N_COL/2,N_LIN, qtdDeParesGH,subSeq_d,auxMatriz_d,i);
   }

   for(i=0; i<N_GPU; i++){
   	   hipMemcpy(subSeq_h + ((BLOCK_SIZE_CALC*N_THREAD_PER_BLOCK_CALC)*i), subSeq_d, (BLOCK_SIZE_CALC*N_THREAD_PER_BLOCK_CALC) * sizeof(int), hipMemcpyDeviceToHost);
   }

   hipEventRecord(stop, 0);
   hipEventSynchronize(stop);
   hipEventElapsedTime(&elapsedTime, start, stop);

   //printf("\n Tempo do kernel (ms) = \%f\n\n", elapsedTime);


   //Encontrando a maior subSeq
   int maiorSubSeq = subSeq_h[0];
   for(i=0; i < ((BLOCK_SIZE_CALC*N_THREAD_PER_BLOCK_CALC)*N_GPU); i++){
	   maiorSubSeq = subSeq_h[i] > maiorSubSeq ? subSeq_h[i]:maiorSubSeq;
	   //printf("%d ", subSeq_h[i]);
   }

  printf("\n Maior SubSequencia encontrada \n\n %d \n\n",maiorSubSeq);

return 0;
}
