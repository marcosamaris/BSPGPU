#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define N 50000//450000000//157370000//524288//157370000//1310720//262144//131072//262144//83886080

//Quantidade de threads por blocos
#define BLOCK_SIZE 1//1024//32//95536
#define nThreadsPerBlock 128//128//420 ou 416

#define nGPU 1
#define nVetorFinal 
#define nVetFinalGPUs ((BLOCK_SIZE * 128) * 5) 

/*
	6.14 versao com coalesced e thread's auxiliando na copia de memoria

	Instancia de 157370000

	com 420 threads

*/

__device__ void memoria(int *vetDados,int *vetComp, int ElemPorBlocos, int qtdProces){

	int aux = (qtdProces * 4096);

	int comecoBloco = blockIdx.x * ElemPorBlocos; // onde cada bloco irá comeca

	int idCompartilhada = threadIdx.x;
	int idGlobal = threadIdx.x + aux + comecoBloco;

	int i;
	for(i = 0; i < 4096; i += blockDim.x){
		vetComp[idCompartilhada] = vetDados[idGlobal];
		idCompartilhada += blockDim.x;
		idGlobal += blockDim.x;
	}
}

__global__ void subSeqMax(int *vet, int *vetFinal, int ElemPorThread, int n){	
		
	__shared__ int p[4096];
	
	//      M         t_m       S      suf
	int ini_M, fim_M, t_M, ini_S, fim_S, suf; //Variaveis do algoritmo
	t_M = suf = 0;

	int comecoThread = (threadIdx.x * 32);

	int j;
	for(j = 0; j < (n / 4096); j++){ // Quantas vezes terei que processa até chegar no n/blocos sendo que o vet compartilhado é de 4076

		memoria(vet,p,n,j);

		__syncthreads();

			if(threadIdx.x < 128){

				ini_M = fim_M = ini_S = fim_S = comecoThread -1;

				int i;
				for(i = comecoThread -1; i < comecoThread + 32; i++){
					if(i == fim_M){
				    	fim_S++;
				    	suf += p[i+1];

				    	if(suf < 0){
				        	suf = 0;
				        	fim_S = -1;
				   		 }
				    
						ini_S = fim_S == 0 ? 0 : ini_S; // Inicio S

				     	 if(p[i+1] > 0){
				           fim_M++;
				           t_M += p[i+1];
				           ini_M = fim_M == 0 ? 0 : ini_M; // Inicio M
				       	 }
					}
					else{
						 if(suf + p[i+1] > t_M){
						     fim_S++;
						     if(ini_M == -1){
						      fim_S = ini_S = i +1;
						     }

						     suf += p[i+1];
						     ini_M = ini_S;
						     fim_M = fim_S;
						     t_M = suf;
						 }
						 else{
						        if(suf + p[i+1] > 0){
						            fim_S++;
						            if(suf == 0){
						                ini_S = fim_S = i+1;
						            }

						            suf += p[i+1];
						        }
						        else{
						            ini_S = fim_S = i + 2;
						            suf = 0;
						        }
				     	}//else
					}//else
				}// 1* for
		}// If 128	
	}// 2* for

	if(threadIdx.x < 128){
		int idThread = blockIdx.x * blockDim.x + threadIdx.x;

		vetFinal[(idThread * 5)] =  vetFinal[(idThread * 5)+1] = vetFinal[(idThread * 5)+2] = vetFinal[(idThread * 5)+3] =
		vetFinal[(idThread * 5)+4] = -1;

		//Colocando o M
		vetFinal[(idThread * 5)+2] = t_M;

		//Calculando o Prefixo
		int pref_Max, soma_Pref;
		soma_Pref  = 0;
		pref_Max = 0;

		int i;
		if(ini_M > comecoThread -1){
		    for(i = 0; i < ini_M; i++){
		        soma_Pref += p[i];

		        if(soma_Pref > pref_Max){
		            pref_Max = soma_Pref;
		        }
		    }

		    if(pref_Max == 0){
				vetFinal[(idThread * 5)] = 0;
				vetFinal[(idThread * 5)+1] = soma_Pref;
		    }
		    else{
				vetFinal[(idThread * 5)] = pref_Max; //Prefixo
				vetFinal[(idThread * 5)+1] = soma_Pref - pref_Max; //Numeros negativos
		    }
		}

		//Calculo do sufixo
		int suf_Max, soma_Suf;
		soma_Suf = suf_Max = 0;

		if(fim_M < comecoThread + 32){
		    for(i = (comecoThread + 32)-1; i > fim_M; i--){
		        soma_Suf += p[i];

		        if(soma_Suf > suf_Max){
		            suf_Max = soma_Suf;
		        }
		    }

		    if(suf_Max == 0){
		        vetFinal[(idThread * 5)+3] = 0;	//Sufixo vazio
				vetFinal[(idThread * 5)+4] = suf_Max;//Os Numeros negativos

		    }
		    else{
		        vetFinal[(idThread * 5)+3] = suf_Max;	//Sufixo vazio
				vetFinal[(idThread * 5)+4] = soma_Suf - suf_Max;//Os Numeros negativos
		    }
		}
	}//if 128
}

void subSeqMaxFinal(int *vet, int n){

    //      M         t_m       S      suf
    int ini_M, fim_M, t_M, ini_S, fim_S, suf;
    ini_M = fim_M = ini_S = fim_S = -1;

    t_M = suf = 0;

	int start;
	int tmili;
	start = clock();


	int i;
    for(i = -1; i < n-1; i++){
        if(i == fim_M){
            fim_S++;
            suf += vet[i+1];

            if(suf < 0){
                suf = 0;
                fim_S = -1;
            }

            ini_S = fim_S == 0 ? 0 : ini_S; // Inicio S

              if(vet[i+1] > 0){
                   fim_M++;
                   t_M += vet[i+1];
                   ini_M = fim_M == 0 ? 0 : ini_M; // Inicio M
               }
        }
        else{
             if(suf + vet[i+1] > t_M){
                 fim_S++;
                 if(ini_M == -1){
                  fim_S = ini_S = i +1;
                 }

                 suf += vet[i+1];
                 ini_M = ini_S;
                 fim_M = fim_S;
                 t_M = suf;

             }
             else{
                    if(suf + vet[i+1] > 0){
                        fim_S++;
                        if(suf == 0){
                            ini_S = fim_S = i+1;
                        }

                        suf += vet[i+1];

                    }
                    else{
                        ini_S = fim_S = i + 2;
                        suf = 0;
                    }
             }
        }
    }

	tmili = (int)((clock()-start)*1000/CLOCKS_PER_SEC);

	printf(" \n\n A sub Sequencia deu %d  \n\n", t_M);

	printf("Tempo total do sequencial %d  \n\n",tmili);

}

int main(){

	float elapsedTime;    // Tempo
	hipEvent_t start, stop; // Tempo

	//Vetores que serao usado no device 
	int *vet_d; // Vetor de dados device
	int *vetFinalGPUs_d; // Vetor final que as GPUs iram devolver

	//Vetores que serao usado no host
	int *vet_h = (int *) malloc(sizeof(int) * N); // Vetor Dados
	int *vetFinal_h = (int *) malloc (sizeof(int) * (nVetFinalGPUs * nGPU));// Vetor Final, as GPU iram devolver o seu vetor final dentro dle

	int i;
	for(i = 0; i < N; i++){ // Preenchimento dos dados
	     vet_h[i] = -1;
	}

	for(i = 0; i < (nVetFinalGPUs * nGPU); i++){ // Preenchimento dos dados
	     vetFinal_h[i] = -1;
	}

	vet_h[70] = 300;

//-----------------------------------------------Reservando espaço de memória e copiando os dados para o device
																													
	for(i = 0; i < nGPU; i++){
		hipSetDevice(i);        //(N / nGPU) porque cada GPU vai cuida de intervalo da sequencia original

		hipMalloc((void**)&vet_d, (N / nGPU) * sizeof(int)); //Vetor de dados em cada GPU
		hipMalloc((void**)&vetFinalGPUs_d, nVetFinalGPUs * sizeof(int)); // Vetor final que cada GPU irá receber

    	hipMemcpy(vet_d, (vet_h) + ((N / nGPU) * i),  (N / nGPU) * sizeof(int), hipMemcpyHostToDevice);
						//(vet_h) + ((N / nGPU) * i) falo a onde esta cada intervalo das GPU
	}

//------------------------------------------------------------------------------------------------------Kernels

	int ElemPorGPU = N / nGPU; // Cada GPU fica responsavel por um intervalo de N/nGPU do vetor original
	int ElemPorBlocos = (ElemPorGPU / BLOCK_SIZE); //Cada bloco fica responsavel por um intervalo da sequencia origina
	int ElemPorThread = (ElemPorBlocos / nThreadsPerBlock); //Cada thread fica responsavel por um intervalo da sequencia original
	
	hipEventCreate(&start); // Contagem do tempo
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	for(i = 0; i < nGPU; i++){
		hipSetDevice(i);
		subSeqMax<<<BLOCK_SIZE, nThreadsPerBlock>>>(vet_d, vetFinalGPUs_d, ElemPorThread, ElemPorBlocos);
	}


	hipDeviceSynchronize();

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);



//	printf("Primeiro kernel (ms) = \%f\n\n", elapsedTime);

	
//-----------------------------------------------------------------------------------------------------------------
	
	for(i = 0; i < nGPU; i++){
		hipSetDevice(i);
		hipMemcpy(vetFinal_h + (i * nVetFinalGPUs) , vetFinalGPUs_d, nVetFinalGPUs * sizeof(int), hipMemcpyDeviceToHost); //Resposta Final
	}

//---------------------------------------------------------------------------------------Formando o vetor Final



	for(i = 0; i < 4096; i++){
		if(vetFinal_h[i] != 0 && vetFinal_h[i] != -1 )
			printf("%d ", vetFinal_h[i]);	
	}
		
	printf("\n\n");

	hipFree(vetFinalGPUs_d);
	hipFree(vet_d);

    	subSeqMaxFinal(vetFinal_h, (nVetFinalGPUs * nGPU));

	return 0;
}

